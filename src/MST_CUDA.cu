
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

constexpr int MAX_THREADS = (1 << 9);

struct edge {
    int u, v, w;
    edge(int u, int v, int w): u(u), v(v), w(w) {}
};

using comparison_func_t = bool (*) (edge*, edge*);

int n;
edge *edges, *chosen_edges;
int *par;
int num_edge;

__device__ bool comparison_weight(edge *x, edge *y) {
    if (x->w == y->w) {
        if (x->u == y->u)
            return x->v < y->v;
        return x->u < y->u;
    }
    return x->w < y->w;
}

__device__ bool comparison_node(edge *x, edge *y) {
    if (x->u == y->u)
        return x->v < y->v;
    return x->u < y->u;
}

__device__ comparison_func_t p_comparison_weight = comparison_weight;
__device__ comparison_func_t p_comparison_node = comparison_node;

int get_container_length(int x) {
    int ret = 1;
    while (ret < x)
        ret <<= 1;
    return ret;
}

__global__ void bitonic_sort_kernel(edge *d_edges, int j, int k, comparison_func_t comparison) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    auto swap = [&](edge& x, edge& y)->void {
        edge temp = x;
        x = y;
        y = temp;
    };

    if (ixj > i) {
        if (((i & k) != 0) && (*comparison)(&d_edges[i], &d_edges[ixj]))
            swap(d_edges[i], d_edges[ixj]);
        else if (((i & k) == 0) && (*comparison)(&d_edges[ixj], &d_edges[i]))
            swap(d_edges[i], d_edges[ixj]);
    }
}

void bitonic_sort(edge *edges, int length, comparison_func_t comparison) {
    int container_length = get_container_length(length);
    for (int i = length; i < container_length; i++) {
        edges[i] = edge(INT_MAX, INT_MAX, INT_MAX);
    }
    length = container_length;

    edge *d_edges;
    size_t container_size = length * sizeof(edge);

    // Copy data to gpu
    hipMalloc((void**) & d_edges, container_size);
    hipMemcpy(d_edges, edges, container_size, hipMemcpyHostToDevice);

    // Call kernel func
    int num_thread = min(length, MAX_THREADS);
    int num_blocks = length / num_thread;
    dim3 blocks(num_blocks, 1);
    dim3 threads(num_thread, 1);

    for (int k = 2; k <= length; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            bitonic_sort_kernel<<<blocks, threads>>>(d_edges, j, k, comparison);
        }
    }

    // Copy result from gpu
    hipMemcpy(edges, d_edges, container_size, hipMemcpyDeviceToHost);
    hipFree(d_edges);
}

int main(int argc, char **argv) {
    // Copy function to device
    comparison_func_t h_comparison_weight;
    comparison_func_t h_comparison_node;

    hipMemcpyFromSymbol(&h_comparison_weight, HIP_SYMBOL(p_comparison_weight), sizeof(comparison_func_t));
    hipMemcpyFromSymbol(&h_comparison_node, HIP_SYMBOL(p_comparison_node), sizeof(comparison_func_t));

    // Init clock
    clock_t t = clock();

    // Input n
    cin >> n;

    // Initialize parents
    par = (int * ) malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        par[i] = i;
    }

    function<int(int)> find_set = [&](int x) {
        return (par[x] == x ? x : par[x] = find_set(par[x]));
    };

    function<bool(int, int)> merge_set = [&](int u, int v) {
        int pu = find_set(u), pv = find_set(v);
        if (pu == pv) return false;
        par[pv] = pu;
        return true;
    };

    // Input edge
    edges = (edge * ) malloc(n * n * sizeof(edge));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int x;
            cin >> x;
            if (x == -1 || i >= j) continue;
            edges[num_edge++] = edge(i, j, x);
        }
    }
    assert(num_edge >= n - 1);

    // Sort weight
    bitonic_sort(edges, num_edge, h_comparison_weight);

    // Build MST
    long long total_cost = 0;
    int num_chosen = 0;
    chosen_edges = (edge * ) malloc(num_edge * 2 * sizeof(edge));
    for (int i = 0; i < num_edge; i++) {
        int u = edges[i].u, v = edges[i].v, w = edges[i].w;
        if (merge_set(u, v)) {
            total_cost += w;
            chosen_edges[num_chosen++] = edges[i];
            if (num_chosen == n - 1) break;
        }
    }

    // Sort chosen edge for output
    bitonic_sort(chosen_edges, num_chosen, h_comparison_node);

    // Get duration
    double time_taken = ((double) (clock() - t)) / CLOCKS_PER_SEC;

    // Output
    cout << total_cost << '\n';
    for (int i = 0; i < num_chosen; i++) {
        cout << chosen_edges[i].u << '-' << chosen_edges[i].v << '\n';
    }
    cout << fixed << setprecision(12) << "Waktu eksekusi: " << time_taken << " ms\n";

    // Return
    return 0;
}
